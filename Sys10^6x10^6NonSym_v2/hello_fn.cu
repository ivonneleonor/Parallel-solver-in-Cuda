#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include "hipblas.h"
#include "hello.cuh"

#define imin(a,b) (a<b?a:b)
#define CLEANUP(s)      \
do{                     \
   printf("%s\n",s);    \
   if(I)     free(I);   \
   if(J)     free(J);   \
   if(val)   free(val); \
   if(r0)     free(r0);   \
   if(csrRowPtrA)  hipFree(csrRowPtrA);   \
   if(csrColIndA)  hipFree(csrColIndA);   \
   if(valA)  hipFree(valA);   \
   if(x)    hipFree(x);     \
   if(r)    hipFree(r);     \
   if(d_Ax)   hipFree(d_Ax);    \
   if(descrA)          hipsparseDestroyMatDescr(descrA);  \
   if(cublasHandle)   hipblasDestroy(cublasHandle);   \
   if(handle) hipsparseDestroy(handle);   \
   hipDeviceReset(); \
   fflush(stdout); \
} while(0)

/*
__global__ void set(double *dx,int N)
{
 int tid=threadIdx.x+blockIdx.x*blockDim.x ;
 if (tid<N)
 dx[tid]=0.0;
}
*/

extern "C"

{
 double *solverbicg(int* J,double* val,int* I,double* x0,double* r0, int N,int nz)
{

//const size_t sz=size_t(N)*sizeof(int);
/*
FILE *p1;
p1=fopen("x0.dat","w+");
FILE *p2;
p2=fopen("datosiniciales.dat","w+");
FILE *p3;
p3=fopen("csrRowPtrA.dat","w+");
FILE *p4;
p4=fopen("csrColIndA.dat","w+");
FILE *p5;
p5=fopen("d_valsILU0.dat","w+");
*/
FILE *p6;
p6=fopen("Ax.dat","w+");
//FILE *p7;
//p5=fopen("d_valsILU0.dat","w+");
//const int threadsPerBlock = 1024;
//const int blocksPerGrid = imin( 32, (N+threadsPerBlock-1) / threadsPerBlock );
int i,k;  
const float tol = 1e-16f;
const int maxit = 1;
double alpha=1.0,  beta=0.0,  alfa=0.0,alphan=0.0, rho=0.0, rhop=0.0, temp=0,temp2=0, nrmr=0, nrmr0=0, uno=1.0, cero=0.0, omega=1.0, omegan=0.0;
double *d_Ax=0, *test=0,*test1=0, *valA=0,*x=0, *r=0, *d_valsILU0=0, *valL=0, *valU=0, *f=0, *p=0, *rw=0, *t=0, *ph=0, *q=0, *s=0;
int *csrRowPtrA=0, *csrColIndA=0, *csrRowPtrL=0, *csrRowPtrU=0, *csrColIndL=0, *csrColIndU=0;
// *test2=0, *test3=0;
int nzILU0 = 2*N-1;
hipError_t cudaStat1,cudaStat2,cudaStat3, cudaStat4,cudaStat5,cudaStat6,cudaStat7, cudaStat8, cudaStat9, cudaStat10,cudaStat11, cudaStat12, cudaStat13;
hipblasStatus_t cublasStatus1;
hipblasHandle_t cublasHandle=0;
hipsparseStatus_t cusparseStatus1;
hipsparseHandle_t handle;
hipsparseMatDescr_t descrA;

cusparseStatus1=hipsparseCreate(&handle);
 if(cusparseStatus1!=HIPSPARSE_STATUS_SUCCESS){
   CLEANUP("Cusparse create handle failed\n");
 }

cublasStatus1=hipblasCreate(&cublasHandle);
  if(cublasStatus1!=HIPBLAS_STATUS_SUCCESS){
     CLEANUP("Cublas create handle failed \n");}


cusparseStatus1=hipsparseCreateMatDescr(&descrA);
  if(cusparseStatus1!=HIPSPARSE_STATUS_SUCCESS){
        printf("Descriptor creation failed\n");
 }//Set matrix type and index base

hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL);
hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO);


test = (double *)malloc(sizeof(double)*nz);
test1 = (double *)malloc(sizeof(double)*N);
//test2 = (int *)malloc(sizeof(int )*N);
//test3 = (int *)malloc(sizeof(int )*nz);
if((!test)||(!test1)){CLEANUP("Memory on host failed,test\n");}

/*
   for(i=0;i<nz;i++){
       fprintf(p2,"i=%d,J=%d,val=%7.3f,I=%d, r=%7.3f \n",i,J[i],val[i],I[i],r0[i]);}
   printf("N=%d  nz=%d\n",N,nz);
*/


    cudaStat1=hipMalloc((void **)&csrRowPtrA, (N+1)*sizeof(int));
    cudaStat2=hipMalloc((void **)&csrColIndA, nz*sizeof(int));
    cudaStat3=hipMalloc((void **)&valA, nz*sizeof(double));
    cudaStat4=hipMalloc((void **)&x, N*sizeof(double));
    cudaStat5=hipMalloc((void **)&r, N*sizeof(double));
    cudaStat7=hipMalloc((void **)&d_Ax, N*sizeof(double));
    cudaStat8=hipMalloc((void **)&csrRowPtrL, (N+1)*sizeof(int));
    cudaStat9=hipMalloc((void **)&csrRowPtrU, (N+1)*sizeof(int));
    cudaStat10=hipMalloc((void **)&csrColIndL, nz*sizeof(int));
    cudaStat11=hipMalloc((void **)&csrColIndU, nz*sizeof(int));
    cudaStat12=hipMalloc((void **)&f, N*sizeof(double));
    if((cudaStat1!=hipSuccess)||(cudaStat2!=hipSuccess)||(cudaStat3!=hipSuccess)||(cudaStat4!=hipSuccess)||(cudaStat5!=hipSuccess)||(cudaStat7!=hipSuccess)||(cudaStat8!=hipSuccess)||(cudaStat9!=hipSuccess)||(cudaStat10!=hipSuccess)||(cudaStat11!=hipSuccess)||(cudaStat12!=hipSuccess)){printf("allocate memory on device"); }
    
//set initial values
    cudaStat1=hipMemcpy(csrColIndA, J, nz*sizeof(int), hipMemcpyHostToDevice);
    cudaStat2=hipMemcpy(csrRowPtrA, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    cudaStat3=hipMemcpy(valA, val, nz*sizeof(double), hipMemcpyHostToDevice);
    cudaStat4=hipMemcpy(x, x0, N*sizeof(double), hipMemcpyHostToDevice);
    cudaStat5=hipMemcpy(f, r0, N*sizeof(double), hipMemcpyHostToDevice);
    if((cudaStat1!=hipSuccess)||(cudaStat2!=hipSuccess)||(cudaStat3!=hipSuccess)||(cudaStat5!=hipSuccess)){printf("Memcpy from Host to Device failed\n");
          } 

//   set<<<blocksPerGrid,threadsPerBlock>>>(x,N);
  

//  cudaStat1=hipMemset(x,0,sz);
//if(cudaStat1!=hipSuccess){printf("set x0 to 0 failed\n");
//          }

/*
   cudaStat1=hipMemcpy(test2,x,(N)*sizeof(int),hipMemcpyDeviceToHost);
   if(cudaStat1!=hipSuccess){printf("Memcpy from Host to Device failed\n");
          }
printf("x de la copia \n");

   for(i=0;i<N;i++)
    {
      fprintf(p1,"%d\n",test2[i]);
    }
*/


 // Preconditioned Conjugate Gradient using ILU.
 
    cudaStat1=hipMalloc((void **)&d_valsILU0, nz*sizeof(double));
    cudaStat2=hipMalloc((void **)&q, N*sizeof(double));
    cudaStat7=hipMalloc((void **)&valL, nz*sizeof(double));
    cudaStat8=hipMalloc((void **)&valU, nz*sizeof(double));
    cudaStat9=hipMalloc((void **)&p, N*sizeof(double));
    cudaStat10=hipMalloc((void **)&rw, N*sizeof(double));
    cudaStat11=hipMalloc((void **)&t, N*sizeof(double));
    cudaStat12=hipMalloc((void **)&ph, N*sizeof(double));
    cudaStat13=hipMalloc((void **)&s, N*sizeof(double));
    if((cudaStat1!=hipSuccess)||(cudaStat2!=hipSuccess)||(cudaStat7!=hipSuccess)||(cudaStat8!=hipSuccess)||(cudaStat9!=hipSuccess)||(cudaStat10!=hipSuccess)||(cudaStat11!=hipSuccess)||(cudaStat12!=hipSuccess)||(cudaStat13!=hipSuccess)){printf("allocate memory on device 2"); }


/* create the analysis info object for the A matrix */
    cusparseSolveAnalysisInfo_t infoA = 0;
    cusparseCreateSolveAnalysisInfo(&infoA);

/* Perform the analysis for the Non-Transpose case */
  cusparseStatus1 =cusparseDcsrsv_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                             N, nz, descrA, valA,  csrRowPtrA, csrColIndA, infoA);
    if((cusparseStatus1!=HIPSPARSE_STATUS_SUCCESS)){printf("Dcsrsv_analysis (1)failed\n");}

 /* Copy A data to ILU0 vals as input*/
   cudaStat1=hipMemcpy(d_valsILU0, valA, nz*sizeof(double), hipMemcpyDeviceToDevice);
   if((cudaStat1!=hipSuccess)){printf("hipMemcpyDeviceToDevice (2) failed \n");}

    /* generate the Incomplete LU factor H for the matrix A using cudsparseScsrilu0 */
  cusparseStatus1=cusparseDcsrilu0(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, descrA, d_valsILU0,csrRowPtrA, csrColIndA,  infoA);if(cusparseStatus1!=HIPSPARSE_STATUS_SUCCESS){printf("hipMemcpyDeviceToDevice (3) failed\n");}
/*
cudaStat1=hipMemcpy(test2,csrRowPtrA, (N+1)*sizeof(int), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" d_row hipMemcpyDeviceToHost failed\n");}
    printf(" d_row \n ");
    for(i=0;i<(N+1);i++){
     fprintf(p3,"%d\t",test2[i]);}
     printf("\n");

cudaStat1=hipMemcpy(test3,csrColIndA, nz*sizeof(int), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" d_row hipMemcpyDeviceToHost failed\n");}
    printf(" d_col \n ");
    for(i=0;i<nz;i++){
     fprintf(p4,"%d\t",test3[i]);}
     printf("\n");

cudaStat1=hipMemcpy(test,d_valsILU0, nz*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" d_valsILU0 hipMemcpyDeviceToHost failed\n");}
    printf(" d_valsILU0 \n ");
    for(i=0;i<nz;i++){
     fprintf(p5,"%e\t",test[i]);}
     printf("\n");

*/


   /* Copy ILU0 data to valL and valU*/
    cudaStat1=hipMemcpy(valL, d_valsILU0, nz*sizeof(double), hipMemcpyDeviceToDevice);
    cudaStat2=hipMemcpy(valU, d_valsILU0, nz*sizeof(double), hipMemcpyDeviceToDevice);
    cudaStat3=hipMemcpy(csrRowPtrL,csrRowPtrA, (N+1)*sizeof(int), hipMemcpyDeviceToDevice);
    cudaStat4=hipMemcpy(csrRowPtrU,csrRowPtrA, (N+1)*sizeof(int), hipMemcpyDeviceToDevice);
    cudaStat5=hipMemcpy(csrColIndL,csrColIndA, nz*sizeof(int), hipMemcpyDeviceToDevice);
    cudaStat6=hipMemcpy(csrColIndU,csrColIndA, nz*sizeof(int), hipMemcpyDeviceToDevice);
    if((cudaStat1!=hipSuccess)||(cudaStat2!=hipSuccess)||(cudaStat3!=hipSuccess)||(cudaStat4!=hipSuccess)||(cudaStat5!=hipSuccess)||(cudaStat6!=hipSuccess)){printf("hipMemcpyDeviceToDevice (4) failed\n");}

/*
cudaStat1=hipMemcpy(test2,csrRowPtrL, (N+1)*sizeof(int), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" csrRowPtrL hipMemcpyDeviceToHost  failed\n");}
    printf("  csrRowPtrL \n ");
    for(i=0;i<(N+1);i++){
     printf("%d\t",test2[i]);}
     printf("\n");

cudaStat1=hipMemcpy(test2,csrRowPtrU, (N+1)*sizeof(int), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" csrRowPtrU  failed\n");}
    printf("csrRowPtrU  \n ");
    for(i=0;i<(N+1);i++){
     printf("%d\t",test2[i]);}
     printf("\n");

cudaStat1=hipMemcpy(test3,csrColIndL, nz*sizeof(int), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" csrColIndL hipMemcpyDeviceToHost failed\n");}
    printf(" csrColIndL \n ");
    for(i=0;i<nz;i++){
     printf("%d\t",test3[i]);}
     printf("\n");

cudaStat1=hipMemcpy(test3,csrColIndU, nz*sizeof(int), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" csrColIndU hipMemcpyDeviceToHost failed\n");}
    printf(" csrColIndU \n ");
    for(i=0;i<nz;i++){
     printf("%d\t",test3[i]);}
     printf("\n");


cudaStat1=hipMemcpy(test,valL, nz*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" valL hipMemcpyDeviceToHost failed\n");}
    printf(" valL \n ");
    for(i=0;i<nz;i++){
     printf("%e\t",test[i]);}
     printf("\n");

cudaStat1=hipMemcpy(test,valU, nz*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" valU hipMemcpyDeviceToHost failed\n");}
    printf(" valU \n ");
    for(i=0;i<nz;i++){
     printf("%e\t",test[i]);}
     printf("\n");
*/

   /* Create info objects for the ILU0 preconditioner */
    cusparseSolveAnalysisInfo_t infoU;
    cusparseCreateSolveAnalysisInfo(&infoU);
    cusparseSolveAnalysisInfo_t infoL;
    cusparseCreateSolveAnalysisInfo(&infoL);

    hipsparseMatDescr_t descrL = 0;
    cusparseStatus1=hipsparseCreateMatDescr(&descrL);
     if((cusparseStatus1!=HIPSPARSE_STATUS_SUCCESS)){printf("hipsparseCreateMatDescr(&descrL) (5) failed\n");}
    hipsparseSetMatType(descrL,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrL,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);
    cusparseStatus1=cusparseDcsrsv_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descrL, valL, csrRowPtrL, csrColIndL, infoL); 
    if(cusparseStatus1!=HIPSPARSE_STATUS_SUCCESS){printf("cusparseDcsrsv_analysis (6) failed\n");}
    
    hipsparseMatDescr_t descrU = 0;
    cusparseStatus1=hipsparseCreateMatDescr(&descrU);
    if(cusparseStatus1!=HIPSPARSE_STATUS_SUCCESS){printf("hipsparseCreateMatDescr(&descrU) (7) failed\n");}
    hipsparseSetMatType(descrU,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrU,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
    hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    cusparseStatus1=cusparseDcsrsv_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descrU, valU, csrRowPtrU, csrColIndU, infoU);
    if((cusparseStatus1!=HIPSPARSE_STATUS_SUCCESS)){printf("cusparseDcsrsv_analysis (8) failed\n");}
/*
    cudaStat1=hipMemcpy(test,valA, nz*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" valU hipMemcpyDeviceToHost failed\n");}
    printf(" valA \n ");
    for(i=0;i<nz;i++){
     printf("%e\t",test[i]);}
     printf("\n");

      cudaStat1=hipMemcpy(test2,csrRowPtrA, (N+1)*sizeof(int), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" csrRowPtrU  failed\n");}
    printf("csrRowPtrA  \n ");
    for(i=0;i<(N+1);i++){
     printf("%d\t",test2[i]);}
     printf("\n");

     cudaStat1=hipMemcpy(test3,csrColIndA, nz*sizeof(int), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" csrColIndU hipMemcpyDeviceToHost failed\n");}
    printf(" csrColIndA \n ");
    for(i=0;i<nz;i++){
     printf("%d\t",test3[i]);}
     printf("\n");
    
     printf("alfa=%f \n",alpha);
     printf("beta=%f \n",beta);

  */   
   
    //1:compute initial Ax0 con x0=0     
    cusparseStatus1=hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N,N,nz,&alpha,descrA,valA,csrRowPtrA,csrColIndA,x,&beta,r);
    if(cusparseStatus1!=HIPSPARSE_STATUS_SUCCESS){printf("hipsparseDcsrmv (9) failed\n");}
/*
    cudaStat1=hipMemcpy(test,r, N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP("Memcpy from Device to Host failed(9)\n");}
    printf(" residuo Ax0, \n ");
    for(i=0;i<N;i++){
     printf("%e\t",test[i]);}
     printf("\n");   
*/
//we can avoid this step as r=Ax0=0 as x0=0 
//hipblasDscal(n,-1.0,r,1);
hipblasDaxpy(cublasHandle,N,&alpha,f,1,r,1);
/*
 cudaStat1=hipMemcpy(test,r, N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP("Memcpy from Device to Host failed(9)\n");}
    printf(" r=f=b=vect=b-Ax0, con Ax0=0, \n ");
    for(i=0;i<N;i++){
     printf("%e\t",test[i]);}
     printf("\n");
*/
//2: Set p=r and \tilde{r}=r

hipblasDcopy(cublasHandle, N,r,1,p,1);
/*
cudaStat1=hipMemcpy(test,p, N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP("Memcpy from Device to Host failed(9)\n");}
    printf(" p=r, \n ");
    for(i=0;i<N;i++){
     printf("%e\t",test[i]);}
     printf("\n");
*/
hipblasDcopy(cublasHandle, N,r,1,rw,1);
/*
cudaStat1=hipMemcpy(test,rw, N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP("Memcpy from Device to Host failed(9)\n");}
    printf(" rw=p, \n ");
    for(i=0;i<N;i++){
     printf("%e\t",test[i]);}
     printf("\n");
*/

   hipblasDdot(cublasHandle,N,r,1,r,1,&rho);
   printf("rho=%f \n",rho);
   nrmr0=rho;

   k=1;

//3: repeat until convergence (based on max. it. and relative residual)
     for(i=0;i<maxit;i++)
   {
      
       //4: \rho=\tilde{r}^{T} r
       rhop=rho;
       hipblasDdot(cublasHandle,N,rw,1,r,1,&rho);
     //  printf("rho=%f \n",rho);
       
       if (i>0)
       {
       //12:
       beta=(rho/rhop)*(alpha/omega);
  //     printf("beta=%f \n",beta); 
  
       //13: p=r+\beta (p- \omega v)
       
       hipblasDaxpy(cublasHandle,N,&omegan,q,1,p,1);
/*
      cudaStat1=hipMemcpy(test,p, N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP("p-omega*v failed\n");}
    printf(" p-omega*v \n ");
    for(i=0;i<N;i++){
     printf("%e\t",test[i]);}
     printf("\n");
*/

       hipblasDscal(cublasHandle,N,&beta,p,1);
/*
        cudaStat1=hipMemcpy(test,p, N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP("beta(p-omega*v) failed \n");}
    printf("beta( p-omega*v) \n ");
    for(i=0;i<N;i++){
     printf("%e\t",test[i]);}
     printf("\n");
*/
       hipblasDaxpy(cublasHandle,N,&uno,r,1,p,1);
  /*      
     cudaStat1=hipMemcpy(test,p, N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP("r+beta(p-omega*v) failed \n");}
    printf("r + beta( p-omega*v) \n ");
    for(i=0;i<N;i++){
     printf("%e\t",test[i]);}
     printf("\n");
*/

        }


      //15: M\hat{p}=p sparse lower and upper triangular solves

cusparseDcsrsv_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N,&alpha, descrL, valL, csrRowPtrL, csrColIndL, infoL,p,t);
/*
cudaStat1=hipMemcpy(test,t, N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP("d_ y solve triangular system\n");}
    printf(" t \n ");
    for(i=0;i<N;i++){
     printf("%e\t",test[i]);}
     printf("\n");
*/

   cusparseDcsrsv_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N,&alpha, descrU, valU, csrRowPtrU, csrColIndU, infoU,t,ph);
/*
   cudaStat1=hipMemcpy(test,ph, N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP("ph \n");}
    printf(" ph   \n ");
    for(i=0;i<N;i++){
     printf("%e\t",test[i]);}
     printf("\n");
*/

//16
     hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N,N,nzILU0, &alpha,descrA,valA,csrRowPtrA,csrColIndA,ph,&beta,q);
/*
cudaStat1=hipMemcpy(test,q, N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" q=Aph \n");}
    printf(" q=Ap   \n ");
    for(i=0;i<N;i++){
     printf("%e\t",test[i]);}
     printf("\n");
//17:\alpha=\rho_{i} / (\tilde{r}^{T} q)
*/
   hipblasDdot(cublasHandle,N,rw,1,q,1,&temp);
  //     printf("temp=rw*p=%f \n",temp);

   alfa=rho/temp;
// printf("alpha=%f \n",alfa);
   alphan=-alfa;
//18: s=r -\alpha q
   hipblasDaxpy(cublasHandle,N,&alphan, q,1,r,1);
/*
  cudaStat1=hipMemcpy(test,r,N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" s=r -alpha q failed\n");}
    printf(" s=r -alpha  \n ");
    for(i=0;i<N;i++){
     printf("%e\t",test[i]);}
     printf("\n");
*/
    //19 x=x+\alpha \hat{p}

   hipblasDaxpy(cublasHandle,N,&alfa, ph,1,x,1);
/*
   cudaStat1=hipMemcpy(test,x,N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" x failed\n");}
    printf("x \n ");
    for(i=0;i<N;i++){
     printf("%e\t",test[i]);}
     printf("\n");
*/
//20: check for convergence
    hipblasDnrm2(cublasHandle,N,r,1, &nrmr);
    printf(" nrmr= %f \n", nrmr);
     printf("  nrmr/nrmr0 %f \n", nrmr/nrmr0);
     if(nrmr/nrmr0<tol)
        {break;}
   
    
//23:  M\hat{s}=p sparse lower and upper triangular solves
     cusparseDcsrsv_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N,&uno, descrL, valL, csrRowPtrL, csrColIndL, infoL,r,t);
  /*   
     cudaStat1=hipMemcpy(test,t,N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP(" t failed\n");}
    printf("t M*hat{s}=r \n ");
    for(i=0;i<N;i++){
     printf("%e\t",test[i]);}
     printf("\n");
*/

     cusparseDcsrsv_solve(handle, HIPSPARSE_OPERATION_TRANSPOSE, N,&uno, descrU, valU, csrRowPtrU, csrColIndU, infoU,t,s);

/*
    cudaStat1=hipMemcpy(test,s,N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP("s failed\n");}
    printf("s M*hat{s}=r \n ");
    for(i=0;i<N;i++){
     printf("%e\t",test[i]);}
     printf("\n");
*/
    //24:t=A\hat{s} (sparse matrix-vector multiplication)

    hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,N,N,nzILU0,&uno,descrA, valA, csrRowPtrA,csrColIndA,s,&cero,t);
/*
    cudaStat1=hipMemcpy(test,t,N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP("t failed\n");}
    printf("t=A*hat{s} \n ");
    for(i=0;i<N;i++){
     printf(" %e\t",test[i]);}
     printf("\n");
  */
     //25 \omega=
    hipblasDdot(cublasHandle,N,t,1,r,1,&temp);
    printf("temp=%f \n",temp);

    hipblasDdot(cublasHandle,N,t,1,t,1,&temp2);
    printf("temp2=%f \n",temp2);


    omega=temp/temp2;
   // printf("omega=%f \n",omega); 

    //26:
    hipblasDaxpy(cublasHandle,N,&omega,s,1,x,1);
/*
    cudaStat1=hipMemcpy(test,x,N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP("x_i failed\n");}
    printf("x_i \n ");
    for(i=0;i<N;i++){
     printf(" %e\t",test[i]);}
     printf("\n");
*/
    omegan=-omega;

    hipblasDaxpy(cublasHandle,N,&omegan,t,1,r,1);
/*
    cudaStat1=hipMemcpy(test,r,N*sizeof(double), hipMemcpyDeviceToHost);
        if(cudaStat1!=hipSuccess){
           CLEANUP("r_i failed\n");}
    printf("r_i \n ");
    for(i=0;i<N;i++){
     printf(" %e\t",test[i]);}
     printf("\n");
*/
   
//20: check for convergence
    hipblasDnrm2(cublasHandle,N,r,1, &nrmr);
    printf(" nrmr= %f \n", nrmr);
     printf("  nrmr/nrmr0 %f \n", nrmr/nrmr0);
     if(nrmr/nrmr0<tol)
        {break;}

   hipDeviceSynchronize();

    k++;


   }

   
     printf("residuo=%e, iteraciones=%d \n",nrmr/nrmr0,k);


    alpha=1.0;
    beta=0.0;

 hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,N,N,nzILU0,&uno,descrA, valA, csrRowPtrA,csrColIndA,s,&cero,t);


cusparseStatus1=hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descrA, valA, csrRowPtrA,csrColIndA, x, &beta, d_Ax);
    if(cusparseStatus1!=HIPSPARSE_STATUS_SUCCESS){
    CLEANUP("Ax0 performing failed\n");}


  cudaStat1=hipMemcpy(test, d_Ax, N*sizeof(double), hipMemcpyDeviceToHost);
  if(cudaStat1!=hipSuccess){
    CLEANUP("Memcpy from Device to Host failed\n"); }
  printf(" d_Ax_(j+1), \n ");
  for(i=0;i<N;i++){
     fprintf(p6,"%e\n",test[i]);
    // test3[i]=test2[i];
  }
  printf("\n");











//printf("iteration = %3d, residual = %e\n", k, r1);

  //  fclose(p1);
  //  fclose(p2);
  //  fclose(p3);
  //  fclose(p4);
    fclose(p6);
    /* Destroy paramters */
    cusparseDestroySolveAnalysisInfo(infoA);
    cusparseDestroySolveAnalysisInfo(infoU);
    cusparseDestroySolveAnalysisInfo(infoL);

    /* Destroy contexts */

    hipsparseDestroy(handle);
    hipblasDestroy(cublasHandle);

    free(I);
    free(J);
    free(val);
    free(r0);
    free(test);
    hipFree(csrRowPtrA);
    hipFree(csrColIndA);
    hipFree(csrRowPtrL);
    hipFree(csrColIndL);
    hipFree(csrRowPtrU);
    hipFree(csrColIndU);
    hipFree(valA);
    hipFree(x);
    hipFree(r);
    hipFree(d_Ax);
    hipFree(valL);
    hipFree(valU);
    hipFree(t);
    hipFree(ph);


    hipDeviceReset();

 return test1;

}
}
